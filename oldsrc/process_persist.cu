#include "hip/hip_runtime.h"
/*
 * process_persist.cu
 *
 * Performs processing by copying data from host to device memory
 * and then running a persistence-based gapfill.
 *
 * All variables in GPU device memory are suffixed with '_d'
 *
 * tim.burgess@noaa.gov 
 */


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "gapfill.h"


// error check
void checkCUDAError(const char *msg) {

  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }  
  
}

// report on available device memory
void checkCUDAMemory() {

  size_t freeMem, totalMem;
  hipMemGetInfo(&freeMem, &totalMem);
  printf("total: %ld Kbytes free: %ld Kbytes\n", totalMem/1024, freeMem/1024);

}

// init workarea  kernel
__global__ void init_workarea(float *workarea) {

  int index = 1024 * blockIdx.x + threadIdx.x;
  *(workarea + index) = -3.0;
}


// gapfill kernel
__global__ void gapfill(short *target, float *workarea, unsigned short* sst, unsigned char* quality,
             unsigned char *lmask) {
  
  int index = 1024 * blockIdx.x + threadIdx.x;
    
  // in landmask, 0 is land and 1 is ocean
  if (*(lmask+index) == 0) { // for land, set to -3
    *(target+index) = 0;
  }
  else { // if quality is > 3, update sst value in workarea
    if (*(quality+index) > 3) {
      *(workarea+index) = (*(sst+index))*0.075 - 3.0;
    }

    // simply load existing workarea value into target
    *(target+index) = (short) ((*(workarea+index)) + 3) * 100;
     
  }

}

extern unsigned char *landmask;
extern short *target;

// use device constant memory for landmask
__device__ __constant__ unsigned char *landmask_d;
__device__ unsigned short *sst_d;
__device__ unsigned char *quality_d;
__device__ float *workarea_d;
__device__ short *target_d;

// memory sizes
size_t sstMemSize, qualityMemSize, targetMemSize;

void preprocess() {

  // report on free mem
  checkCUDAMemory();

  // allocate device memory for landmask
  size_t landMemSize = NPIXELS * sizeof(char);
  hipMalloc((void **)&landmask_d, landMemSize);
  printf("allocating %ld device Kbytes for landmask\n", landMemSize/1024);
  checkCUDAError("memory allocation");
    
  // copy landmask from host to device
  hipMemcpy(landmask_d, landmask, landMemSize, hipMemcpyHostToDevice);
  checkCUDAError("memory copy");
 
  // allocate device memory for sst
  sstMemSize = NPIXELS * sizeof(unsigned short);
  hipMalloc((void **)&sst_d, sstMemSize);
  printf("allocating %ld device Kbytes for sst\n", sstMemSize/1024);
  checkCUDAError("memory allocation");

  // allocate device memory for quality 
  qualityMemSize = NPIXELS * sizeof(char);
  hipMalloc((void **)&quality_d, qualityMemSize);
  printf("allocating %ld device Kbytes for quality\n", qualityMemSize/1024);
  checkCUDAError("memory allocation");

  // allocate memory for workarea
  size_t workMemSize = NPIXELS * sizeof(float);
  hipMalloc((void **)&workarea_d, workMemSize);
  printf("allocating %ld device Kbytes for workarea\n", workMemSize/1024);

  // init workarea values to -3
  init_workarea <<<32768, 1024>>> (workarea_d);

  // allocate memory for target sst
  targetMemSize = NPIXELS * sizeof(short);
  target = (short*)malloc(targetMemSize);
  hipMalloc((void **)&target_d, targetMemSize);
  printf("allocating %ld device Kbytes for target sst\n", targetMemSize/1024);
}

// main
void process(unsigned short* sst, unsigned char* quality) {
 
  // copy sst from host to device
  hipMemcpy(sst_d, sst, sstMemSize, hipMemcpyHostToDevice);
  checkCUDAError("memory copy");
    
  // copy quality from host to device
  hipMemcpy(quality_d, quality, qualityMemSize, hipMemcpyHostToDevice);
  checkCUDAError("memory copy");

  // do calc on device
  //dim3 dimGrid(1024,128);
  //dim3 dimBlock(16,16);
  gapfill <<<32768, 1024>>> (target_d, workarea_d, sst_d, quality_d, landmask_d);

  // block until the device has completed
  hipDeviceSynchronize();
  
  // check for errors
  checkCUDAError("kernel invocation");
  
  // retrieve target data from device
  hipMemcpy(target, target_d, targetMemSize, hipMemcpyDeviceToHost);  
  
}

// wrapup after processing
void postprocess() {

  free(target);
  hipFree(landmask_d);
  hipFree(sst_d);
  hipFree(quality_d);
  hipFree(workarea_d);
  hipFree(target_d);

}
