#include "hip/hip_runtime.h"
/*
 * process_decay.cu
 *
 * Performs processing by copying data from host to device memory
 * and then running an exponential decay-based gapfill.
 *
 * All variables in GPU device memory are suffixed with '_d'
 *
 * tim.burgess@noaa.gov 
 */


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "gapfill.h"

#define SEA_AIR_DELTA 0.75
#define DECAY 0.887

#define THREADS_PER_BLOCK 512


// error check
void checkCUDAError(const char *msg) {

  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }  
  
}

// report on available device memory
void checkCUDAMemory() {

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  printf("-----------------------\n");
  printf("Name: %s\n", prop.name);
  printf("Compute capability: %d.%d\n", prop.major, prop.minor);
  printf("Total Global Memory: %ld Kbytes\n", prop.totalGlobalMem/1024);
  printf("Total Constant Memory: %ld Kbytes\n", prop.totalConstMem/1024);
  printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
  printf("-----------------------\n");
  
  //size_t freeMem, totalMem;
  //hipMemGetInfo(&freeMem, &totalMem);

}

// init workarea  kernel
__global__ void init_workarea(float *workarea, unsigned short *daycount, float *airtemp) {

  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index < NPIXELS) {
    *(workarea + index) = -3.0;
    *(daycount + index) = 0;
    *(airtemp + index) = -3.0;
  }
}


// gapfill kernel
__global__ void gapfill(short *target, float *workarea, unsigned short *sst, unsigned char *quality,
             unsigned char *lmask, unsigned short *daycount, float *airtemp, int indexcount) {
  
  int index = (blockDim.x * blockIdx.x + threadIdx.x) + indexcount;

  if (index < NPIXELS) { // if valid index
    
    // in landmask, 0 is land and 1 is ocean
    if (*(lmask+index) == 0) { // for land, set to -3
      *(target+index) = 0;
    }
    else { // if quality is > 3, update sst value in workarea
      if (*(quality + index) > 3) {
        *(daycount + index) = 0;
        *(workarea + index) = (*(sst + index)) * 0.075 - 3.0;
        *(airtemp + index) = (*(workarea + index)) * SEA_AIR_DELTA;
      }
      else {
        if (*(airtemp + index) != -3) { // no valid calc if a retrieval has not yet occurred
          ++*(daycount + index);
          *(workarea + index) = *(airtemp + index) + SEA_AIR_DELTA * __expf(-1 * (*(daycount + index)))/DECAY;
        }
      }

      // load workarea value into target
      *(target + index) = (short) (*(workarea + index) + 3) * 100;
    }
  }
}

extern unsigned char *landmask;
extern short *target;

// use device constant memory for landmask
__device__ __constant__ unsigned char *landmask_d;
__device__ unsigned short *sst_d;
__device__ unsigned char *quality_d;
__device__ unsigned short *daycount_d;
__device__ float *airtemp_d;
__device__ float *workarea_d;
__device__ short *target_d;

// memory sizes
size_t sstMemSize, qualityMemSize, targetMemSize;

void preprocess() {

  // report on free mem
  checkCUDAMemory();

  // allocate device memory for landmask
  size_t landMemSize = NPIXELS * sizeof(char);
  hipMalloc((void **)&landmask_d, landMemSize);
  printf("allocating %ld device Kbytes for landmask\n", landMemSize/1024);
  checkCUDAError("memory allocation");
    
  // copy landmask from host to device
  hipMemcpy(landmask_d, landmask, landMemSize, hipMemcpyHostToDevice);
  checkCUDAError("memory copy");
 
  // allocate device memory for sst
  sstMemSize = NPIXELS * sizeof(unsigned short);
  hipMalloc((void **)&sst_d, sstMemSize);
  printf("allocating %ld device Kbytes for sst\n", sstMemSize/1024);
  checkCUDAError("memory allocation");

  // allocate device memory for quality 
  qualityMemSize = NPIXELS * sizeof(char);
  hipMalloc((void **)&quality_d, qualityMemSize);
  printf("allocating %ld device Kbytes for quality\n", qualityMemSize/1024);
  checkCUDAError("memory allocation");

  // allocate memory for daycount
  size_t dayMemSize = NPIXELS * sizeof(short);
  hipMalloc((void **)&daycount_d, dayMemSize);
  printf("allocating %ld device Kbytes for daycount\n", dayMemSize/1024);

  // allocate memory for airtemp
  size_t airMemSize = NPIXELS * sizeof(float);
  hipMalloc((void **)&airtemp_d, airMemSize);
  printf("allocating %ld device Kbytes for airtemp\n", airMemSize/1024);

  // allocate memory for workarea
  size_t workMemSize = NPIXELS * sizeof(float);
  hipMalloc((void **)&workarea_d, workMemSize);
  printf("allocating %ld device Kbytes for workarea\n", workMemSize/1024);

  // init working values 
  init_workarea <<<32768, 1024>>> (workarea_d, daycount_d, airtemp_d);

  // allocate memory for target sst
  targetMemSize = NPIXELS * sizeof(short);
  target = (short*)malloc(targetMemSize);
  hipMalloc((void **)&target_d, targetMemSize);
  printf("allocating %ld device Kbytes for target sst\n", targetMemSize/1024);
}

// main
void process(unsigned short* sst, unsigned char* quality) {
 
  // copy sst from host to device
  hipMemcpy(sst_d, sst, sstMemSize, hipMemcpyHostToDevice);
  checkCUDAError("memory copy");
    
  // copy quality from host to device
  hipMemcpy(quality_d, quality, qualityMemSize, hipMemcpyHostToDevice);
  checkCUDAError("memory copy");

  // run gafill twice due to memory constraints
  //gapfill <<<32768, 1024>>> (target_d, workarea_d, sst_d, quality_d, landmask_d, daycount_d, airtemp_d);
  gapfill <<<32768, THREADS_PER_BLOCK>>> (target_d, workarea_d, sst_d, quality_d, landmask_d, daycount_d, airtemp_d, 0);
  gapfill <<<32768, THREADS_PER_BLOCK>>> (target_d, workarea_d, sst_d, quality_d, landmask_d, daycount_d, airtemp_d, NPIXELS/2);

  // block until the device has completed
  hipDeviceSynchronize();
  
  // check for errors
  checkCUDAError("kernel invocation");
  
  // retrieve target data from device
  hipMemcpy(target, target_d, targetMemSize, hipMemcpyDeviceToHost);  
  
}

// wrapup after processing
void postprocess() {

  free(target);
  hipFree(landmask_d);
  hipFree(sst_d);
  hipFree(quality_d);
  hipFree(daycount_d);
  hipFree(airtemp_d);
  hipFree(workarea_d);
  hipFree(target_d);

}



  
